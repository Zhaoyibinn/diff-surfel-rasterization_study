/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

#define CHECK_INPUT(x)											\
	AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
	// AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
	auto lambda = [&t](size_t N) {
		t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
	};
	return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,//背景颜色
	const torch::Tensor& means3D,//三维点的三维坐标
	const torch::Tensor& colors,//在python中为colors_precomp，输入不是RGB则必须，单纯渲染的时候是空
	const torch::Tensor& opacity,//三维点的不透明度
	const torch::Tensor& scales,//scale参数，三个方向的尺度
	const torch::Tensor& rotations,//rotation参数，高斯的旋转，四元数
	const float scale_modifier,//应该是尺度参数，默认为1
	const torch::Tensor& transMat_precomp,
	const torch::Tensor& viewmatrix,//相机外参矩阵(观察矩阵)
	const torch::Tensor& projmatrix,//global投影矩阵，内参矩阵和外参矩阵一通计算得到
	const float tan_fovx, //视野角（单侧）
	const float tan_fovy,//视野角（单侧）
	const int image_height,//图片尺寸
	const int image_width,//图片尺寸
	const torch::Tensor& sh,//全部的球谐系数
	const int degree,//球谐系数的层数，默认3，四层
	const torch::Tensor& campos,//相机的光心，用位姿反算出来的
	const bool prefiltered,//默认false
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
	AT_ERROR("means3D must have dimensions (num_points, 3)");
  }

  
  const int P = means3D.size(0);//多少个点
  const int H = image_height;
  const int W = image_width;

  CHECK_INPUT(background);
  CHECK_INPUT(means3D);
  CHECK_INPUT(colors);
  CHECK_INPUT(opacity);
  CHECK_INPUT(scales);
  CHECK_INPUT(rotations);
  CHECK_INPUT(transMat_precomp);
  CHECK_INPUT(viewmatrix);
  CHECK_INPUT(projmatrix);
  CHECK_INPUT(sh);
  CHECK_INPUT(campos);

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);
//分别存储了int和float类型的三维点坐标信息
  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor out_others = torch::full({3+3+1, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
//   开辟空间
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
	  }

	  rendered = CudaRasterizer::Rasterizer::forward(//光栅化前向通路
		geomFunc,
		binningFunc,
		imgFunc,
		P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		transMat_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		out_others.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, out_others, radii, geomBuffer, binningBuffer, imgBuffer);
//   最后返回给python的就是这个tuple
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
	 const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
	const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& transMat_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
	const torch::Tensor& dL_dout_color,
	const torch::Tensor& dL_dout_others,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug) 
{

  CHECK_INPUT(background);
  CHECK_INPUT(means3D);
  CHECK_INPUT(radii);
  CHECK_INPUT(colors);
  CHECK_INPUT(scales);
  CHECK_INPUT(rotations);
  CHECK_INPUT(transMat_precomp);
  CHECK_INPUT(viewmatrix);
  CHECK_INPUT(projmatrix);
  CHECK_INPUT(sh);
  CHECK_INPUT(campos);
  CHECK_INPUT(binningBuffer);
  CHECK_INPUT(imageBuffer);
  CHECK_INPUT(geomBuffer);

  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dnormal = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dtransMat = torch::zeros({P, 9}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 2}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  transMat_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dout_others.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dnormal.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dtransMat.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dtransMat, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}
